#include "hip/hip_runtime.h"
#include "intersections.h"

__host__ __device__ float boxIntersectionTest(
    Geom box,
    Ray r,
    glm::vec3& intersectionPoint,
    glm::vec3& normal,
    bool& outside)
{
    Ray q;
    q.origin = multiplyMV(box.inverseTransform, glm::vec4(r.origin, 1.0f));
    q.direction = glm::normalize(multiplyMV(box.inverseTransform, glm::vec4(r.direction, 0.0f)));

    float tmin = -1e38f;
    float tmax = 1e38f;
    glm::vec3 tmin_n;
    glm::vec3 tmax_n;
    for (int xyz = 0; xyz < 3; ++xyz)
    {
        float qdxyz = q.direction[xyz];
        /*if (glm::abs(qdxyz) > 0.00001f)*/
        {
            float t1 = (-0.5f - q.origin[xyz]) / qdxyz;
            float t2 = (+0.5f - q.origin[xyz]) / qdxyz;
            float ta = glm::min(t1, t2);
            float tb = glm::max(t1, t2);
            glm::vec3 n;
            n[xyz] = t2 < t1 ? +1 : -1;
            if (ta > 0 && ta > tmin)
            {
                tmin = ta;
                tmin_n = n;
            }
            if (tb < tmax)
            {
                tmax = tb;
                tmax_n = n;
            }
        }
    }

    if (tmax >= tmin && tmax > 0)
    {
        outside = true;
        if (tmin <= 0)
        {
            tmin = tmax;
            tmin_n = tmax_n;
            outside = false;
        }
        intersectionPoint = multiplyMV(box.transform, glm::vec4(getPointOnRay(q, tmin), 1.0f));
        normal = glm::normalize(multiplyMV(box.invTranspose, glm::vec4(tmin_n, 0.0f)));
        return glm::length(r.origin - intersectionPoint);
    }

    return -1;
}

__host__ __device__ float sphereIntersectionTest(
    Geom sphere,
    Ray r,
    glm::vec3& intersectionPoint,
    glm::vec3& normal,
    bool& outside)
{
    float radius = .5;

    glm::vec3 ro = multiplyMV(sphere.inverseTransform, glm::vec4(r.origin, 1.0f));
    glm::vec3 rd = glm::normalize(multiplyMV(sphere.inverseTransform, glm::vec4(r.direction, 0.0f)));

    Ray rt;
    rt.origin = ro;
    rt.direction = rd;

    float vDotDirection = glm::dot(rt.origin, rt.direction);
    float radicand = vDotDirection * vDotDirection - (glm::dot(rt.origin, rt.origin) - powf(radius, 2));
    if (radicand < 0)
    {
        return -1;
    }

    float squareRoot = sqrt(radicand);
    float firstTerm = -vDotDirection;
    float t1 = firstTerm + squareRoot;
    float t2 = firstTerm - squareRoot;

    float t = 0;
    if (t1 < 0 && t2 < 0)
    {
        return -1;
    }
    else if (t1 > 0 && t2 > 0)
    {
        t = min(t1, t2);
        outside = true;
    }
    else
    {
        t = max(t1, t2);
        outside = false;
    }

    glm::vec3 objspaceIntersection = getPointOnRay(rt, t);

    intersectionPoint = multiplyMV(sphere.transform, glm::vec4(objspaceIntersection, 1.f));
    normal = glm::normalize(multiplyMV(sphere.invTranspose, glm::vec4(objspaceIntersection, 0.f)));
    if (!outside)
    {
        normal = -normal;
    }

    return glm::length(r.origin - intersectionPoint);
}

//From 4610
__host__ __device__ float triangleIntersectionTest(glm::vec3 p0, glm::vec3 p1, glm::vec3 p2, Ray r) {
    const float offset = 0.0000001;
    glm::vec3 edge1, edge2, h, s, q;
    float a, f, u, v;
    edge1 = p1 - p0;
    edge2 = p2 - p0;
    h = cross(r.direction, edge2);
    a = dot(edge1, h);
    if (a > -offset && a < offset) {
        return INFINITY;    // This ray is parallel to this triangle.
    }
    f = 1.0 / a;
    s = r.origin - p0;
    u = f * dot(s, h);
    if (u < 0.0 || u > 1.0)
        return INFINITY;
    q = cross(s, edge1);
    v = f * dot(r.direction, q);
    if (v < 0.0 || u + v > 1.0) {
        return INFINITY;
    }
    // At this stage we can compute t to find out where the intersection point is on the line.
    float t = f * dot(edge2, q);
    if (t > EPSILON) {
        return t;
    }
    else // This means that there is a line intersection but not a ray intersection.
        return -1;
}

__host__ __device__ glm::vec3 barycentric(glm::vec3 p, glm::vec3 t1, glm::vec3 t2, glm::vec3 t3) {
    glm::vec3 edge1 = t2 - t1;
    glm::vec3 edge2 = t3 - t2;
    float S = length(cross(edge1, edge2));

    edge1 = p - t2;
    edge2 = p - t3;
    float S1 = length(cross(edge1, edge2));

    edge1 = p - t1;
    edge2 = p - t3;
    float S2 = length(cross(edge1, edge2));

    edge1 = p - t1;
    edge2 = p - t2;
    float S3 = length(cross(edge1, edge2));

    return glm::vec3(S1 / S, S2 / S, S3 / S);
}

__host__ __device__ float meshIntersectionTest(Geom mesh, Ray r, glm::vec3& intersectionPoint, glm::vec3& normal, bool& outside, const Triangle* triangles, glm::vec2& uv) {
    // Transform the ray into object space
    Ray localRay;
    localRay.origin = glm::vec3(mesh.inverseTransform * glm::vec4(r.origin, 1.0f));
    localRay.direction = glm::normalize(glm::vec3(mesh.inverseTransform * glm::vec4(r.direction, 0.0f)));

    float t_min = INFINITY;
    glm::vec3 tmp_intersect, tmp_normal;
    glm::vec2 tmp_uv;

    // Iterate over the triangles in the mesh
    for (int i = mesh.triIndexStart; i < mesh.triIndexEnd; ++i) {
        const Triangle& tri = triangles[i];

        // Perfrom tri ray-triangle intersection for each triangle
        float t = triangleIntersectionTest(tri.v0, tri.v1, tri.v2, localRay);

        // Update closest intersection
        if (t < t_min && t > 0.0f) {
            t_min = t;
            tmp_intersect = getPointOnRay(localRay, t);
            tmp_normal = glm::normalize(glm::cross(tri.v1 - tri.v0, tri.v2 - tri.v0));
            //check if this correct
            glm::vec3 bary = barycentric(tmp_intersect, tri.v0, tri.v1, tri.v2);
            tmp_uv = bary.x * tri.uv0 + bary.y * tri.uv1 + bary.z * tri.uv2;
        }
    }

    // If no intersection was found, transform the point and normal back to world space
    if (t_min < INFINITY) {
        intersectionPoint = multiplyMV(mesh.transform, glm::vec4(tmp_intersect, 1.0f));
        normal = glm::normalize(multiplyMV(mesh.invTranspose, glm::vec4(tmp_normal, 0.0f)));
        uv = tmp_uv;
        return t_min;
    }

    // No intersection
    return -1.0f;
}